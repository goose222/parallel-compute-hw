#include "hip/hip_runtime.h"

#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#define TILE_WIDTH 16


// __global__ static void MatrixMulKernel(const float* Md,const float* Nd,float* Pd,int Width)
__global__ void MatrixMulKernel(const float* Md, const float* Nd, float* Pd, int x, int y, int z, float gamma)
{
    float Pvalue = 0.0;
    //计算Pd和Md中元素的行索引
    int Row = blockIdx.x * TILE_WIDTH + threadIdx.x; //行
    int Col = blockIdx.y * TILE_WIDTH + threadIdx.y; //列

    __shared__ float a_share[TILE_WIDTH][TILE_WIDTH];
    __shared__ float b_share[TILE_WIDTH][TILE_WIDTH];

    for(int i=0; i<y/TILE_WIDTH;i++){
        a_share[threadIdx.x][threadIdx.y]=Md[Row*y+(i*TILE_WIDTH+threadIdx.y)];
        b_share[threadIdx.y][threadIdx.x]=Nd[Col*y+(i*TILE_WIDTH+threadIdx.x)];

        __syncthreads();

        for(int k=0; k<TILE_WIDTH; k++){
            float tmp = (a_share[threadIdx.x][k]-b_share[threadIdx.y][k]);
            Pvalue += pow(tmp,2.0);
        }
        __syncthreads();
    }

    Pd[Row * z + Col] = exp(-1.0 * Pvalue/(2.0 * pow(gamma,2.0)));
}

void MatrixMultiplication_CUDA(const float* M, const float* N, float* P, int x, int y, int z ,float gamma)
{
    hipSetDevice(0);  //设置目标GPU

    float *Md, *Nd, *Pd;
    int size1 = x * y * sizeof(float);//字节长度
    int size2 = y * z * sizeof(float);//字节长度
    int size3 = x * z * sizeof(float);//字节长度

    hipMalloc((void**)&Md, size1);
    hipMalloc((void**)&Nd, size2);
    hipMalloc((void**)&Pd, size3);

    //Copies a matrix from the memory* area pointed to by src to the memory area pointed to by dst
    hipMemcpy(Md, M, size1, hipMemcpyHostToDevice);
    hipMemcpy(Nd, N, size2, hipMemcpyHostToDevice);

    //
    dim3 dimGrid(x / TILE_WIDTH, z / TILE_WIDTH);	//网格的维度
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);					//块的维度
    MatrixMulKernel <<< dimGrid, dimBlock >>>(Md, Nd, Pd, x, y, z, gamma);

    hipMemcpy(P, Pd, size3, hipMemcpyDeviceToHost);
    //释放设备上的矩阵
    hipFree(Md);
    hipFree(Nd);
    hipFree(Pd);
}

void matgen(float* a, int x, int y)
{
    int i, j;
    for (i = 0; i < x; i++)
    {
        for (j = 0; j < y; j++)
        {
            a[i * x + j] = (float)rand() / RAND_MAX + (float)rand() / (RAND_MAX*RAND_MAX);
        }
    }
}

void MatrixMultiplication(const float* M, const float* N, float* P, int x, int y, int z ,float gamma)//x*y  & y*z
{
    int i, j, k;
    for (i = 0; i < x; i++)
    {
        for (j = 0; j < z; j++)
        {
            float sum = 0;
            for (k = 0; k < y; k++)
            {
                sum += pow((M[i * x + k] - N[k * z + j]),2);
            }
            P[i * x + j] = exp(-1.0 * sum/(2.0 * pow(gamma,2.0)));
        }
    }
}

double MatrixMul_GPU()
{
    float *M, *N, *Pg;
    int x = 1024;	//1024×1024矩阵乘法
    int y = 1024;
    int z = 1024;
    int gamma = 1;
    M = (float*)malloc(sizeof(float)* x * y);
    N = (float*)malloc(sizeof(float)* y * z);
    Pg = (float*)malloc(sizeof(float)* x * z); //保存GPU计算结果

    srand(0);

    matgen(M, x, y);			//产生矩阵M
    matgen(N, y, z);			//产生矩阵N

    double timeStart, timeEnd;	//定义时间，求时间差用
    timeStart = clock();
    MatrixMultiplication_CUDA(M, N, Pg, x, y, z, gamma);			//GPU上计算
    timeEnd = clock();

    free(M);
    free(N);
    free(Pg);
    return timeEnd - timeStart;
}

double MatrixMul_CPU()
{
    float *M, *N, *Pc;
    int Width = 1024;	//1024×1024矩阵乘法
    int x = 1024;	//1024×1024矩阵乘法
    int y = 1024;
    int z = 1024;
    float gamma = 1;
    M = (float*)malloc(sizeof(float)* x * y);
    N = (float*)malloc(sizeof(float)* y * z);
    Pc = (float*)malloc(sizeof(float)* x * z); //保存GPU计算结果

    srand(0);

    matgen(M, x, y);			//产生矩阵M
    matgen(N, y, z);			//产生矩阵N

    double timeStart, timeEnd; //定义时间，求时间差用
    timeStart = clock();
    MatrixMultiplication(M, N, Pc, x, y, z, gamma);				//CPU上计算
    timeEnd = clock();

    free(M);
    free(N);
    free(Pc);
    return timeEnd - timeStart;
}

int main()
{
    printf("CPU use time %g\n", MatrixMul_CPU());
    printf("GPU use time %g\n", MatrixMul_GPU());
    system("pause");
    return 0;
}